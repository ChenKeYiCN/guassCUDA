#include "hip/hip_runtime.h"
//
// Created by chenk on 2024/12/29.
//
#include "guass.cuh"
#include <cstdio>

cv::Mat guassCPU(const cv::Mat &src, const cv::Size &guass_size, double sigmaX, double sigmaY)
{
    cv::Mat blur;
    GaussianBlur(src, blur, guass_size, sigmaX, sigmaY);
    return blur;
}

cv::Size createKernelSizeBysigma(int imageType, cv::Size ksize, double sigma1, double sigma2)
{
    cv::Size output_size = ksize;
    int depth = CV_MAT_DEPTH(imageType);
    if( sigma2 <= 0 )
        sigma2 = sigma1;
    // automatic detection of kernel size from sigma
    if( ksize.width <= 0 && sigma1 > 0 )
        output_size.width = cvRound(sigma1*(depth == CV_8U ? 3 : 4)*2 + 1)|1;
    if( ksize.height <= 0 && sigma2 > 0 )
        output_size.height = cvRound(sigma2*(depth == CV_8U ? 3 : 4)*2 + 1)|1;

    CV_Assert( output_size.width  > 0 && output_size.width  % 2 == 1 &&
               output_size.height > 0 && output_size.height % 2 == 1 );
    return output_size;
}

void guassGPU(unsigned char *input, unsigned char *output, size_t numRows, size_t numCols, int kernelRowSize,
    int kernelColSize, float *kernelRow, float *kernelCol)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((numCols + threadsPerBlock.x - 1) / threadsPerBlock.x, (numRows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    GaussianKernel1d_row<<<blocksPerGrid, threadsPerBlock>>>(input, output, numRows, numCols, kernelRowSize, kernelRow);
    GaussianKernel1d_col<<<blocksPerGrid, threadsPerBlock>>>(output, input, numRows, numCols, kernelColSize, kernelCol);
}

__global__ void GaussianKernel1d_row(unsigned char *src, unsigned char *dst, int height, int width, int filterWidth, float *filter)
{
    int y = blockDim.y * blockIdx.y + threadIdx.y; //二维块的当前行
    int x = blockDim.x * blockIdx.x + threadIdx.x; //二维块的当前列
    int ind = y * width + x;
    if (y >= height || x >= width)
    {
        return;
    }
    float color = 0.0f;
    int padding = filterWidth / 2;

    for (int i = 0; i < filterWidth; i++)
    {
        float k = filter[i];
        // printf("wtf i=%d, width=%d, k=%f\n", i, filterWidth, k);
        int origin_x = x + i - padding;
        if(origin_x < 0)
            origin_x = std::abs(origin_x) - 1;
        else if(origin_x >= width)
            origin_x = 2 * width - origin_x - 1;

        float s = (float)src[y * width + origin_x];
        color += k * s;
    }
    dst[ind] = min(max((int)color, 0), 255);
}

__global__ void GaussianKernel1d_col(unsigned char *src, unsigned char *dst, int height, int width, int filterWidth, float *filter)
{
    int y = blockDim.y * blockIdx.y + threadIdx.y; //二维块的当前行
    int x = blockDim.x * blockIdx.x + threadIdx.x; //二维块的当前列
    int ind = y * width + x;
    if (y >= height || x >= width)
    {
        return;
    }
    float color = 0.0f;
    int padding = filterWidth / 2;

    for (int i = 0; i < filterWidth; i++)
    {
        float k = filter[i];
        int origin_y = y + i - padding;
        if(origin_y < 0)
            origin_y = std::abs(origin_y) - 1;
        else if(origin_y >= height)
            origin_y = 2 * height - origin_y - 1;

        float s = (float)src[origin_y * width + x];
        color += k * s;
    }
    // dst[x * width + y] = min(max((int)color, 0), 255);
    dst[ind] = min(max((int)color, 0), 255);
    //    dst[ind] = color;
}

